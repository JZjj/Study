#include "hip/hip_runtime.h"
#include "helper.hpp"
const int M          = 32; // the height of the output matrix
const int N          = 32; // the width of the output matrix
const int K          = 4;  // the length of the intermediate dimension in A*B
const int UNROLL     = 32 / K;
const int BLOCKDIM_Y = M / UNROLL;
const int BLOCKDIM_X = N;
const int BLOCKDIM_Z = 1;

// Sequential code for the forward path of the convolution layer
// You should not modify this code
static void conv_forward_valid(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims) {
  std::fill(Y, Y + ydims.flattened_length(), 0);

  for (auto i : range(0, ydims.num)) {
    for (auto m : range(0, ydims.depth)) {    // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width)) {
          const auto yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth)) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {  // filter height
              for (auto q : range(0, wdims.width)) { // filter width
                const auto xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const auto woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Baseline GPU kernel code for forward convolution.
// One thread per output index
// You should not modify this kernel as it is used for correctness comparison.
// Instead, define a new one below
__global__ void conv_forward_baseline_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
                                             const shape ydims) {

  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = gx; i < ydims.num * ydims.depth * ydims.height * ydims.width; i += blockDim.x * gridDim.x) {
    Y[i] = 0.f;
  }

  for (size_t i = gx; i < ydims.num; i += gridDim.x * blockDim.x) {
    for (auto m : range(0, ydims.depth)) {    // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width)) {
          const size_t yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth)) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {  // filter height
              for (auto q : range(0, wdims.width)) { // filter width
                const size_t xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const size_t woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_baseline(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims) {

  dim3 dimGrid(1);
  dim3 dimBlock(32);

  conv_forward_baseline_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());
}

// Implement your optimized kernel here.
// Make any modifications you wish.
// Don't forget to modify the host code below, if needed!
__global__ void conv_forward_opt_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y, const shape ydims) {

  // X[b, c, h+p, w+q] = X[((b * xdims.depth + c) * xdims.height + (h + p)) * xdims.width + (w + q)]
  // W[m, c, p, q] = W[((m * wdims.depth + c) * wdims.height + p) * wdims.width + q]
  // Y[b, m, h, w] = Y[((b * ydims.depth + m) * ydims.height + h) * ydims.width + w]
  const int b     = ydims.num;
  const int wh    = wdims.height;
  const int ww    = wdims.width;
  const int h_in  = ydims.height + wh - 1;
  const int w_in  = ydims.width + ww - 1;
  const int c     = xdims.depth;
  const int m     = ydims.depth;
  const int h_out = h_in - wh + 1;
  const int w_out = w_in - ww + 1;

  int idxXBlockCol = blockIdx.x * m + threadIdx.x;
  int idxM         = blockIdx.y * N + threadIdx.y;

  int numXBlockRows = c * wh * ww;
  int numXBlockCols = h_out * w_out;

  __shared__ float blockMemA[M][K];
  __shared__ float blockMemB[K][N];

  int idxB    = blockIdx.z * BLOCKDIM_Z + threadIdx.z;
  int idxHOut = idxXBlockCol / w_out;
  int idxWOut = idxXBlockCol % w_out;

  if (idxB < b) {
    float PValue[UNROLL];
    for (int unroll = 0; unroll < UNROLL; unroll++) {
      PValue[unroll] = 0.0;
    }

    for (int idx = 0; idx < (numXBlockRows + K - 1) / K; idx++) {
// simplify the matrix indexing
#define Y4d(i3, i2, i1, i0) Y[(i3) * (m * h_out * w_out) + (i2) * (h_out * w_out) + (i1) * (w_out) + i0]
#define W4d(i3, i2, i1, i0) X[(i3) * (c * h_in * w_in) + (i2) * (h_in * w_in) + (i1) * (w_in) + i0]
#define X4d(i3, i2, i1, i0) W[(i3) * (c * wh * ww) + (i2) * (wh * ww) + (i1) * (ww) + i0]

      const int threadIdx1D      = threadIdx.y * BLOCKDIM_X + threadIdx.x;
      const int threadNumInBlock = BLOCKDIM_X * BLOCKDIM_Y;
      int max                    = (M * K + threadNumInBlock - 1) / threadNumInBlock;
      if (((K * N + threadNumInBlock - 1) / threadNumInBlock) > max)
        max = (K * N + threadNumInBlock - 1) / threadNumInBlock;
      for (int idxLoad = 0; idxLoad < max; idxLoad++) {
        int idxYBlockMemALoad = (threadIdx1D + idxLoad * threadNumInBlock) / K;
        int idxXBlockMemALoad = (threadIdx1D + idxLoad * threadNumInBlock) % K;
        int idxYBlockMemBLoad = (threadIdx1D + idxLoad * threadNumInBlock) / N;
        int idxXBlockMemBLoad = (threadIdx1D + idxLoad * threadNumInBlock) % N;

        int idxWColumnToLoad   = idx * K + idxXBlockMemALoad;
        int idxXBlockColToLoad = blockIdx.x * N + idxXBlockMemBLoad;
        int idxXBlockRowToLoad = (idxYBlockMemBLoad + idx * K);

        int idxHOutToLoad = idxXBlockColToLoad / w_out;
        int idxWOutToLoad = idxXBlockColToLoad % w_out;

        int idxCWColumn   = idxWColumnToLoad / (wh * ww);
        int idxMToLoad    = blockIdx.y * M + idxYBlockMemALoad;
        int idxCXBlockRow = idxXBlockRowToLoad / (wh * ww);
        if ((idxCWColumn < c) && (idxMToLoad < m)) {
          int idxKHeightWColumn                           = (idxWColumnToLoad % (wh * ww)) / ww;
          int idxKWidthWColumn                            = (idxWColumnToLoad % (wh * ww)) % ww;
          blockMemA[idxYBlockMemALoad][idxXBlockMemALoad] = X4d(idxMToLoad, idxCWColumn, idxKHeightWColumn, idxKWidthWColumn);
        } else {
          blockMemA[idxYBlockMemALoad][idxXBlockMemALoad] = 0.0;
        }
        if ((idxCXBlockRow < c) && (idxHOutToLoad < h_out)) {
          int idxKHeightXBlockRow = (idxXBlockRowToLoad % (wh * ww)) / ww;
          int idxKWidthXBlockRow  = (idxXBlockRowToLoad % (wh * ww)) % ww;
          blockMemB[idxYBlockMemBLoad][idxXBlockMemBLoad] =
              W4d(idxB, idxCXBlockRow, idxHOutToLoad + idxKHeightXBlockRow, idxWOutToLoad + idxKWidthXBlockRow);
        } else {
          blockMemB[idxYBlockMemBLoad][idxXBlockMemBLoad] = 0.0;
        }
      }
      __syncthreads();
      for (int idx2 = 0; idx2 < K; idx2++) {
        for (int unroll = 0; unroll < UNROLL; unroll++) {
          PValue[unroll] += blockMemA[threadIdx.y + unroll * BLOCKDIM_Y][idx2] * blockMemB[idx2][threadIdx.x];
        }
      }
      __syncthreads();
    }
    if (idxM < M && idxXBlockCol < numXBlockCols) {
      for (int unroll = 0; unroll < UNROLL; unroll++) {
        Y4d(idxB, idxM + unroll * BLOCKDIM_Y, idxHOut, idxWOut) = PValue[unroll];
      }
    }
  }
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_opt(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims) {
  // X[b, c, h+p, w+q] = X[((b * xdims.depth + c) * xdims.height + (h + p)) * xdims.width + (w + q)]
  // W[m, c, p, q] = W[((m * wdims.depth + c) * wdims.height + p) * wdims.width + q]
  // Y[b, m, h, w] = Y[((b * ydims.depth + m) * ydims.height + h) * ydims.width + w]
  const int wh      = wdims.height;
  const int ww      = wdims.width;
  const int m       = ydims.depth;
  const int h_in    = ydims.height + wh - 1;
  const int w_in    = ydims.width + ww - 1;
  const int h_out   = h_in - wh + 1;
  const int w_out   = w_in - ww + 1;
  const int b       = ydims.num;
  int numWtiledRows = m;
  int numXBlockCols = h_out * w_out;
  dim3 blockUnrollDimGolden(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
  dim3 gridUnrollDimGolden((numXBlockCols + BLOCKDIM_X - 1) / BLOCKDIM_X,
                           (numWtiledRows + BLOCKDIM_Y * UNROLL - 1) / (BLOCKDIM_Y * UNROLL),
                           (b + BLOCKDIM_Z - 1) / BLOCKDIM_Z);
  THROW_IF_ERROR(hipMemset(Y, 0, sizeof(float) * ydims.num * ydims.depth * ydims.height * ydims.width));
  conv_forward_opt_kernel<<<gridUnrollDimGolden, blockUnrollDimGolden>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());
}

static int eval(const shape wDims, const shape xDims, bool doVerify) {

  // Generate model
  const auto conf_info = std::string("conv[wDims:") + std::to_string(wDims.num) + "," + std::to_string(wDims.depth) + "," +
                         std::to_string(wDims.height) + "," + std::to_string(wDims.width) + " xDims:" + std::to_string(xDims.num) + "," +
                         std::to_string(xDims.depth) + "," + std::to_string(xDims.height) + "," + std::to_string(xDims.width) + "]";
  INFO("Running " << conf_info);

  // Generate convolution weights
  float *hostW = allocate<float>(wDims);
  generate_convfilters(hostW, wDims);

  // generate input feature map
  float *hostX = allocate<float>(xDims);
  generate_data(hostX, xDims);

  // generate output feature map for verification
  const shape ydims = {xDims.num, wDims.num, (xDims.height - wDims.height + 1), (xDims.width - wDims.width + 1)};
  INFO("Allocating output tensor [" << ydims.num << "," << ydims.depth << "," << ydims.height << "," << ydims.width << "]");
  float *hostY    = allocate<float>(ydims);
  float *expected = allocate<float>(ydims);
  generate_data(hostY, ydims);

  const size_t wByteCount = wDims.flattened_length() * sizeof(float);
  const size_t xByteCount = xDims.flattened_length() * sizeof(float);
  const size_t yByteCount = ydims.flattened_length() * sizeof(float);

  float *deviceW = nullptr, *deviceX = nullptr, *deviceY = nullptr;
  timer_start("Allocating GPU memory.");
  THROW_IF_ERROR(hipMalloc((void **) &deviceW, wByteCount));
  THROW_IF_ERROR(hipMalloc((void **) &deviceX, xByteCount));
  THROW_IF_ERROR(hipMalloc((void **) &deviceY, yByteCount));
  timer_stop();

  timer_start("Copying inputs to the GPU.");
  THROW_IF_ERROR(hipMemcpy(deviceW, hostW, wByteCount, hipMemcpyDefault));
  THROW_IF_ERROR(hipMemcpy(deviceX, hostX, xByteCount, hipMemcpyDefault));
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  convlayer_gpu_opt(deviceX, xDims, deviceW, wDims, deviceY, ydims);
  THROW_IF_ERROR(hipDeviceSynchronize());
  timer_stop();

  // verify with provided implementation
  if (doVerify) {
    timer_start("Copying output to the CPU");
    THROW_IF_ERROR(hipMemcpy(hostY, deviceY, yByteCount, hipMemcpyDefault));
    timer_stop();

    convlayer_gpu_baseline(deviceX, xDims, deviceW, wDims, deviceY, ydims);
    THROW_IF_ERROR(hipDeviceSynchronize());
    THROW_IF_ERROR(hipMemcpy(expected, deviceY, yByteCount, hipMemcpyDefault));
    // conv_forward_valid(hostX, xDims, hostW, wDims, expected, ydims);
    verify(expected, hostY, ydims);
  }

  THROW_IF_ERROR(hipFree(deviceW));
  THROW_IF_ERROR(hipFree(deviceX));
  THROW_IF_ERROR(hipFree(deviceY));
  free(hostW);
  free(hostX);
  free(hostY);
  free(expected);

  return 0;
}

TEST_CASE("Convlayer", "[convlayer]") {
#if 1
  // test five times in case code errors depend on data
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32, 1, 5, 5}, {20, 1, 28, 28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32, 1, 5, 5}, {20, 1, 28, 28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32, 1, 5, 5}, {20, 1, 28, 28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32, 1, 5, 5}, {20, 1, 28, 28}, true);
  }
  SECTION("[wDims:32,1,5,5 xDims:20,1,28,28]") {
    eval({32, 1, 5, 5}, {20, 1, 28, 28}, true);
  }
#else
  SECTION("[wDims:32,1,5,5 xDims:50000,1,28,28]") {
    eval({32, 1, 5, 5}, {50000, 1, 28, 28}, false);
  }
#endif
}

