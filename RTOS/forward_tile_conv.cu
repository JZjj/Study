#include "hip/hip_runtime.h"
const int M          = 32; // the height of the output matrix
const int N          = 32; // the width of the output matrix
const int K          = 4;  // the length of the intermediate dimension in A*B
const int UNROLL     = 32 / K;
const int BLOCKDIM_Y = M / UNROLL;
const int BLOCKDIM_X = N;
const int BLOCKDIM_Z = 1;

__global__ void conv_forward_opt_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y, const shape ydims) {

  // X[b, c, h+p, w+q] = X[((b * xdims.depth + c) * xdims.height + (h + p)) * xdims.width + (w + q)]
  // W[m, c, p, q] = W[((m * wdims.depth + c) * wdims.height + p) * wdims.width + q]
  // Y[b, m, h, w] = Y[((b * ydims.depth + m) * ydims.height + h) * ydims.width + w]
  const int b     = ydims.num;
  const int wh    = wdims.height;
  const int ww    = wdims.width;
  const int h_in  = ydims.height + wh - 1;
  const int w_in  = ydims.width + ww - 1;
  const int c     = xdims.depth;
  const int m     = ydims.depth;
  const int h_out = h_in - wh + 1;
  const int w_out = w_in - ww + 1;

  int idxXBlockCol = blockIdx.x * m + threadIdx.x;
  int idxM         = blockIdx.y * N + threadIdx.y;

  int numXBlockRows = c * wh * ww;
  int numXBlockCols = h_out * w_out;

  __shared__ float blockMemA[M][K];
  __shared__ float blockMemB[K][N];

  int idxB    = blockIdx.z * BLOCKDIM_Z + threadIdx.z;
  int idxHOut = idxXBlockCol / w_out;
  int idxWOut = idxXBlockCol % w_out;

  if (idxB < b) {
    float PValue[UNROLL];
    for (int unroll = 0; unroll < UNROLL; unroll++) {
      PValue[unroll] = 0.0;
    }

    for (int idx = 0; idx < (numXBlockRows + K - 1) / K; idx++) {
// simplify the matrix indexing
#define Y4d(i3, i2, i1, i0) Y[(i3) * (m * h_out * w_out) + (i2) * (h_out * w_out) + (i1) * (w_out) + i0]
#define W4d(i3, i2, i1, i0) X[(i3) * (c * h_in * w_in) + (i2) * (h_in * w_in) + (i1) * (w_in) + i0]
#define X4d(i3, i2, i1, i0) W[(i3) * (c * wh * ww) + (i2) * (wh * ww) + (i1) * (ww) + i0]

      const int threadIdx1D      = threadIdx.y * BLOCKDIM_X + threadIdx.x;
      const int threadNumInBlock = BLOCKDIM_X * BLOCKDIM_Y;
      int max                    = (M * K + threadNumInBlock - 1) / threadNumInBlock;
      if (((K * N + threadNumInBlock - 1) / threadNumInBlock) > max)
        max = (K * N + threadNumInBlock - 1) / threadNumInBlock;
      for (int idxLoad = 0; idxLoad < max; idxLoad++) {
        int idxYBlockMemALoad = (threadIdx1D + idxLoad * threadNumInBlock) / K;
        int idxXBlockMemALoad = (threadIdx1D + idxLoad * threadNumInBlock) % K;
        int idxYBlockMemBLoad = (threadIdx1D + idxLoad * threadNumInBlock) / N;
        int idxXBlockMemBLoad = (threadIdx1D + idxLoad * threadNumInBlock) % N;

        int idxWColumnToLoad   = idx * K + idxXBlockMemALoad;
        int idxXBlockColToLoad = blockIdx.x * N + idxXBlockMemBLoad;
        int idxXBlockRowToLoad = (idxYBlockMemBLoad + idx * K);

        int idxHOutToLoad = idxXBlockColToLoad / w_out;
        int idxWOutToLoad = idxXBlockColToLoad % w_out;

        int idxCWColumn   = idxWColumnToLoad / (wh * ww);
        int idxMToLoad    = blockIdx.y * M + idxYBlockMemALoad;
        int idxCXBlockRow = idxXBlockRowToLoad / (wh * ww);
        if ((idxCWColumn < c) && (idxMToLoad < m)) {
          int idxKHeightWColumn                           = (idxWColumnToLoad % (wh * ww)) / ww;
          int idxKWidthWColumn                            = (idxWColumnToLoad % (wh * ww)) % ww;
          blockMemA[idxYBlockMemALoad][idxXBlockMemALoad] = X4d(idxMToLoad, idxCWColumn, idxKHeightWColumn, idxKWidthWColumn);
        } else {
          blockMemA[idxYBlockMemALoad][idxXBlockMemALoad] = 0.0;
        }
        if ((idxCXBlockRow < c) && (idxHOutToLoad < h_out)) {
          int idxKHeightXBlockRow = (idxXBlockRowToLoad % (wh * ww)) / ww;
          int idxKWidthXBlockRow  = (idxXBlockRowToLoad % (wh * ww)) % ww;
          blockMemB[idxYBlockMemBLoad][idxXBlockMemBLoad] =
              W4d(idxB, idxCXBlockRow, idxHOutToLoad + idxKHeightXBlockRow, idxWOutToLoad + idxKWidthXBlockRow);
        } else {
          blockMemB[idxYBlockMemBLoad][idxXBlockMemBLoad] = 0.0;
        }
      }
      __syncthreads();
      for (int idx2 = 0; idx2 < K; idx2++) {
        for (int unroll = 0; unroll < UNROLL; unroll++) {
          PValue[unroll] += blockMemA[threadIdx.y + unroll * BLOCKDIM_Y][idx2] * blockMemB[idx2][threadIdx.x];
        }
      }
      __syncthreads();
    }
    if (idxM < M && idxXBlockCol < numXBlockCols) {
      for (int unroll = 0; unroll < UNROLL; unroll++) {
        Y4d(idxB, idxM + unroll * BLOCKDIM_Y, idxHOut, idxWOut) = PValue[unroll];
      }
    }
  }
}

static void convlayer_gpu_opt(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y, const shape &ydims) {
  // X[b, c, h+p, w+q] = X[((b * xdims.depth + c) * xdims.height + (h + p)) * xdims.width + (w + q)]
  // W[m, c, p, q] = W[((m * wdims.depth + c) * wdims.height + p) * wdims.width + q]
  // Y[b, m, h, w] = Y[((b * ydims.depth + m) * ydims.height + h) * ydims.width + w]
  const int wh      = wdims.height;
  const int ww      = wdims.width;
  const int m       = ydims.depth;
  const int h_in    = ydims.height + wh - 1;
  const int w_in    = ydims.width + ww - 1;
  const int h_out   = h_in - wh + 1;
  const int w_out   = w_in - ww + 1;
  const int b       = ydims.num;
  int numWtiledRows = m;
  int numXBlockCols = h_out * w_out;
  dim3 blockUnrollDimGolden(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
  dim3 gridUnrollDimGolden((numXBlockCols + BLOCKDIM_X - 1) / BLOCKDIM_X,
                           (numWtiledRows + BLOCKDIM_Y * UNROLL - 1) / (BLOCKDIM_Y * UNROLL),
                           (b + BLOCKDIM_Z - 1) / BLOCKDIM_Z);
  THROW_IF_ERROR(hipMemset(Y, 0, sizeof(float) * ydims.num * ydims.depth * ydims.height * ydims.width));
  conv_forward_opt_kernel<<<gridUnrollDimGolden, blockUnrollDimGolden>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());
}

